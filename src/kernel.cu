#include "hip/hip_runtime.h"
#include "cuda_globals.hpp"
#include "application.hpp"
#include "kernel.cuh"
#include "configuration.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <fstream>
#include <chrono>

#ifndef PI
#define PI 3.14159265359
#endif

// -------------
//  Kernels
// -------------

// for hiprand_uniform (this uses XORWOW and is faster than Philox4)
__global__ void d_setup_kernel(int _n,
                               int _rand_seed,
                               int _rand_offset,
                               hiprandState* _state) {
  int i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _n;
       i += blockDim.x * gridDim.x) {
    hiprand_init(_rand_seed, i, _rand_offset, &_state[i]);
  }
}

// n4 = n/4
__global__ void d_setup_kernel(int _n4,
                               int _rand_seed,
                               int _rand_offset,
                               hiprandStatePhilox4_32_10_t* _state) {
  int i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _n4;
       i += blockDim.x * gridDim.x) {
    hiprand_init(_rand_seed, i, _rand_offset, &_state[i]);
  }
}

template<typename T>
__global__ void d_generate_poisson_numbers(int _n4,
                                           uint4* _poisson_numbers,
                                           T _lambda,
                                           hiprandStatePhilox4_32_10_t* _state) {
  int i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _n4;
       i += blockDim.x * gridDim.x) {
    hiprandStatePhilox4_32_10_t local_state = _state[i];
    _poisson_numbers[i] = hiprand_poisson4(&local_state,
                                          _lambda);
    _state[i] = local_state;
  }
}

template<typename T>
__device__ T genexp(T _lambda, hiprandState* _state) {
  return -1.0/_lambda * log(1.0-hiprand_uniform(_state));
}


template<typename T>
__device__
void d_gen_fischer(hiprandState& _local_state,
                   const Configuration<T>& _params,
                   typename TVec3<T>::type& _orientation
                   // T& beta,
                   // T& lambda
  ) {
  /// gen_fischer
  T waux,a1,a2;
//    int safe = 10000;
  do
  {
    a1=hiprand_uniform(&_local_state);
    a2=hiprand_uniform(&_local_state);
    waux=(1.0-(1.0+_params.fis_b)*a1) / (1.0-(1.0-_params.fis_b)*a1);
    //    if(--safe<0) break;
  }while(_params.fis_k*waux+2.0*log(1.0-_params.fis_x0*waux)-_params.fis_c >= log(a2));

  T u1 = hiprand_uniform(&_local_state);
  T u2 = hiprand_uniform(&_local_state);
  T v1=sqrt(-2*log(u1))*cos(2*PI*u2);
  T v2=sqrt(-2*log(u1))*sin(2*PI*u2);
  v1=v1/(sqrt(v1*v1+v2*v2));
  v2=v2/(sqrt(v1*v1+v2*v2));
  T x = sqrt(1.0-waux*waux)*v1;
  T y = sqrt(1.0-waux*waux)*v2;
  T z = waux;
  if(z<0.0) {
    x = -x; // _orientation
    y = -y;
    z = -z;
  }
  _orientation.x = x;
  _orientation.y = y;
  _orientation.z = z;
  // beta = acos(z); // calculate later
  // lambda = PI+atan2(y, x);
}

template<bool DumpForIntermediates, typename T, typename U>
__global__
void d_compute_fibers(int _nf,
                      typename TVec4<T>::type* _values,
                      U* _fibers_find,
                      const Configuration<T> _config,
                      hiprandState* _states) {

  int i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _nf;
       i += blockDim.x * gridDim.x) {
    hiprandState local_state = _states[i];

    /// position
    T x = _config.xmax * hiprand_uniform(&local_state);
    T y = _config.ymax * hiprand_uniform(&local_state);
    T z = _config.zmax * hiprand_uniform(&local_state);


    /// fiber length
    /// rand_weib(NP,l,m,xc)
    T l = pow(genexp(pow(_config.weib_xc, _config.weib_m), &local_state),
              1.0/_config.weib_m);

    /// fiber orientation
    typename TVec3<T>::type orientation;
    d_gen_fischer(local_state, _config, orientation);

    // if(i==2)
    //   printf("%f %f %f %f\n", z, l, orientation.z, _config.zlevel);

    if(DumpForIntermediates) {
      _values[i].x = x;
      _values[i].y = y;
      _values[i].z = z;
      _values[i].w = l;
      _values[i+_nf].x = orientation.x;
      _values[i+_nf].y = orientation.y;
      _values[i+_nf].z = orientation.z;
      if((z>_config.zlevel) && (z-l*orientation.z)<_config.zlevel) {
        _fibers_find[i] = 1;
      }
    } else {
      if((z>_config.zlevel) && (z-l*orientation.z)<_config.zlevel) {
//      rmin=fmin((z-zlevel)/orientation.z, l-(z-zlevel)/orientation.z); // ??? compute later
//      rmax = l-rmin;
        _values[i].x = x;
        _values[i].y = y;
        _values[i].z = z;
        _values[i].w = l;
        _values[i+_nf].x = orientation.x;
        _values[i+_nf].y = orientation.y;
        _values[i+_nf].z = orientation.z;
        _fibers_find[i] = 1;
        // store rmin, rmax, beta, l, lambda ..?
      }
    }
    _states[i] = local_state;
  }
}

template<typename T>
__device__ inline
T tau(T d) { return 1.02; }

// try to put loop for _d in here
template<typename T, typename TVec4>
__device__ T d_Psingle(const TVec4& _pos,
                       const TVec4& _orientation,
                       T _d,
                       T _ninters,
                       const Configuration<T> _config) {
  T sum = 0;
  T z = _pos.z;
  T l = _pos.w; // fiber length
  T beta = acos(_orientation.z);
//  T lambda = PI + atan2(_orientation.y, _orientation.x);
  T rmin = fmin((z-_config.zlevel)/_orientation.z, l-(z-_config.zlevel)/_orientation.z);
  // rmax = l-rmin; ?
  T Ef, Em, n, u0, sigma0, rhof,aux;
  // @todo use compile-constant expressions?

  Ef= 720.0;
  Em= 500.0;
  n = Ef/Em;
  rhof = _ninters/(4.0/(PI*_config.fdiam*_config.fdiam)+_ninters);
  sigma0 = 4.0*tau(_d)*(rmin)/(_config.fdiam);
  u0  = sigma0*(rmin)/Ef;
  aux = (1.0+n*rhof)/(2.0-n*n*rhof*rhof);

  if (rmin>0 && rmin>_d)
  {
    if(_d<=u0*(1.0+n*rhof))
      sum=PI*_config.fdiam*_config.fdiam/4.0*sigma0*sqrt(_d/u0*(1.0+n*rhof));
    else
      sum=PI*_config.fdiam*_config.fdiam/4.0*sigma0*(rmin/u0*aux-sqrt((rmin/u0*aux)*(rmin/u0*aux)-2.0/u0*(rmin-_d)*aux));
    if( (_d<=u0*(1.0+n*rhof)
       && _d/u0*(1.0+n*rhof) < _config.sigmafu*_config.sigmafu*exp(-2.0*_config.mu*beta))
      || (_d>u0*(1.0+n*rhof) && (sigma0*(1+n*rhof)<_config.sigmafu*exp(-2.0*_config.mu*beta))))
    sum *= exp( _config.mu * beta );
  }
  return sum;
}



// for thrust::copy_if
struct is_flagged
{
  __device__
  bool operator()(const int x) {
    return x==1;
  }
};


template<typename T>
struct ComputeForce {
  using DataVec4T = typename Data<T>::TVec4T;
  const T ninters_;
  const Configuration<T> config_;
  const T dstep_;

  ComputeForce(const T _dstep,
               const int _ninters,
               const Configuration<T> _config)
    : dstep_(_dstep), ninters_(_ninters), config_(_config) {}

  __device__
  T operator()(const DataVec4T& pos, const DataVec4T& orientation) const {
    return d_Psingle(pos, orientation, dstep_, ninters_, config_);
  }
};



// ----------------
//  Host functions
// ----------------


template<typename T>
void cuda_compute_number_of_fibers(const Application<T>& _app,
                                   uint* _output ) {

  Configuration<T> config = _app.configuration();
  dim3 threads(128);
  dim3 blocks( 8*_app.number_sm() );

  int n4 = (config.nr+3) / 4;
  uint4* poisson_numbers_d = nullptr;
  hiprandStatePhilox4_32_10_t* devStatesPhilox = nullptr;
  cuda_malloc(&devStatesPhilox, n4);
  cuda_malloc(&poisson_numbers_d, n4);

  d_setup_kernel<<<blocks, threads>>>(n4,
                                      config.rand_seed,
                                      config.rand_offset,
                                      devStatesPhilox);

  d_generate_poisson_numbers<<<blocks, threads>>>(n4,
                                                  poisson_numbers_d,
                                                  config.lambda,
                                                  devStatesPhilox);
  CHECK_LAST( "Kernel failure.");
  cuda_memcpy(_output, reinterpret_cast<uint*>(poisson_numbers_d), config.nr);
  CHECK_CUDA( hipFree(devStatesPhilox) );
  CHECK_CUDA( hipFree(poisson_numbers_d) );
  poisson_numbers_d = nullptr;


  if(_app.dump_intermediates()) {
    std::ofstream fs_intermediates;
    uint* ptr = _output;
    fs_intermediates.open("_intermediate_numbers_for_np", std::ofstream::out);
    for(int i=0; i<config.nr; ++i) {
      fs_intermediates << i <<", "<< *ptr++ << "\n";
    }
    fs_intermediates.close();
  }

}

template<typename T>
int cuda_create_and_intersect_fibers(const Application<T>& _app,
                                     Data<T>& _data,
                                     int _nr_fibers) {

  Configuration<T> config = _app.configuration();
  dim3 threads(128);
  dim3 blocks( 32*_app.number_sm() );

  hiprandState* devStates = nullptr;
  cuda_malloc(&devStates, _nr_fibers);

  d_setup_kernel<<<blocks, threads>>>(_nr_fibers,
                                      config.rand_seed,
                                      config.rand_offset,
                                      devStates);
  CHECK_LAST( "Kernel failure" );

  // main computation, filters intersecting fibers
  if(_app.dump_intermediates())
    d_compute_fibers<true><<<blocks, threads>>>(_nr_fibers, _data.values, _data.fibers_find, config, devStates);
  else
    d_compute_fibers<false><<<blocks, threads>>>(_nr_fibers, _data.values, _data.fibers_find, config, devStates);
  CHECK_LAST( "Kernel failure" );

  CHECK_CUDA(hipStreamSynchronize(0));

  CHECK_CUDA( hipFree(devStates) );

  int ninters = thrust::reduce(thrust::device,
                               _data.fibers_find,
                               _data.fibers_find + _nr_fibers,
                               0,
                               thrust::plus<int>());
  return ninters;
}


template<typename T>
void cuda_compact_data(Data<T>& _data,
                        int _nr_intersections,
                        int _nr_fibers) {
  cuda_malloc(&_data.values_compact, 2*_nr_intersections);
  cuda_malloc(&_data.forces, _nr_intersections);
  // stream compaction (store only values of intersecting fibers)
  thrust::copy_if(thrust::device,
                  _data.values,
                  _data.values + _nr_fibers,
                  _data.fibers_find,
                  _data.values_compact,
                  is_flagged());
  // second part (orientation, ..)
  thrust::copy_if(thrust::device,
                  _data.values + _nr_fibers,
                  _data.values + 2*_nr_fibers,
                  _data.fibers_find,
                  _data.values_compact + _nr_intersections,
                  is_flagged());
}

template<typename T>
T cuda_compute_force(Data<T>& _data,
                     const Configuration<T>& _config,
                     int _nr_intersections,
                     int _nr_fibers,
                     T _dstep) {
  // inplace transform
  thrust::transform(thrust::device,
                    _data.values_compact,         //first1
                    _data.values_compact + _nr_intersections, //last1
                    _data.values_compact + _nr_intersections, //first2
                    _data.forces,
                    ComputeForce<T>(_dstep, _nr_intersections, _config));

  T value_init = 0;
  T result = thrust::reduce(thrust::device,
                            _data.forces,
                            _data.forces+_nr_intersections,
                            value_init);
  return result;
}



template
void cuda_compute_number_of_fibers<float>(const Application<float>& _app,
                                   uint* _output );

template
int cuda_create_and_intersect_fibers<float>(const Application<float>& _app,
                                     Data<float>& _data,
                                     int _nr_fibers);

template
void cuda_compact_data<float>(Data<float>& _data,
                       int _nr_intersections,
                       int _nr_fibers);


template
float cuda_compute_force<float>(Data<float>& _data,
                     const Configuration<float>& _config,
                     int _nr_intersections,
                     int _nr_fibers,
                     float _dstep);


template
void cuda_compute_number_of_fibers<double>(const Application<double>& _app,
                                   uint* _output );

template
int cuda_create_and_intersect_fibers<double>(const Application<double>& _app,
                                     Data<double>& _data,
                                     int _nr_fibers);

template
void cuda_compact_data<double>(Data<double>& _data,
                       int _nr_intersections,
                       int _nr_fibers);


template
double cuda_compute_force<double>(Data<double>& _data,
                     const Configuration<double>& _config,
                     int _nr_intersections,
                     int _nr_fibers,
                     double _dstep);
